
#include <hip/hip_runtime.h>
// Node data structure matching Rust's NodeData
struct NodeData {
    float position[3];    // 12 bytes
    float velocity[3];    // 12 bytes
    unsigned char mass;   // 1 byte
    unsigned char flags;  // 1 byte
    unsigned char padding[2]; // 2 bytes padding
};

extern "C" __global__ void compute_forces(
    NodeData* nodes,
    int num_nodes,
    float spring_strength,
    float repulsion,
    float damping
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_nodes) return;

    // Load node data
    NodeData node_i = nodes[idx];
    float3 pos_i = make_float3(
        node_i.position[0],
        node_i.position[1],
        node_i.position[2]
    );
    float mass_i = (float)node_i.mass;
    float3 force = make_float3(0.0f, 0.0f, 0.0f);

    __shared__ float3 shared_positions[256];
    __shared__ float shared_masses[256];

    // Process nodes in tiles to maximize shared memory usage
    for (int tile = 0; tile < (num_nodes + blockDim.x - 1) / blockDim.x; tile++) {
        int shared_idx = tile * blockDim.x + threadIdx.x;
        
        // Load tile into shared memory
        if (shared_idx < num_nodes) {
            NodeData shared_node = nodes[shared_idx];
            shared_positions[threadIdx.x] = make_float3(
                shared_node.position[0],
                shared_node.position[1],
                shared_node.position[2]
            );
            shared_masses[threadIdx.x] = (float)shared_node.mass;
        }
        __syncthreads();

        // Compute forces between current node and all nodes in tile
        #pragma unroll 8
        for (int j = 0; j < blockDim.x && tile * blockDim.x + j < num_nodes; j++) {
            if (tile * blockDim.x + j == idx) continue;

            // Skip nodes with inactive flag
            if ((nodes[tile * blockDim.x + j].flags & 0x1) == 0) continue;

            float3 pos_j = shared_positions[j];
            float mass_j = shared_masses[j];
            
            // Calculate displacement vector
            float3 diff = make_float3(
                pos_i.x - pos_j.x,
                pos_i.y - pos_j.y,
                pos_i.z - pos_j.z
            );

            // Calculate force magnitude with minimum distance clamp
            float dist = fmaxf(sqrtf(diff.x * diff.x + diff.y * diff.y + diff.z * diff.z), 0.0001f);
            float force_mag = repulsion * mass_i * mass_j / (dist * dist);

            // Add spring force if nodes are connected (check flags)
            if ((node_i.flags & 0x2) && (nodes[tile * blockDim.x + j].flags & 0x2)) {
                float spring_force = spring_strength * (dist - 1.0f); // Natural length = 1.0
                force_mag += spring_force;
            }

            // Accumulate force
            force.x += force_mag * diff.x / dist;
            force.y += force_mag * diff.y / dist;
            force.z += force_mag * diff.z / dist;
        }
        __syncthreads();
    }

    // Load current velocity
    float3 vel = make_float3(
        node_i.velocity[0],
        node_i.velocity[1],
        node_i.velocity[2]
    );

    // Update velocity with damping
    vel.x = (vel.x + force.x) * damping;
    vel.y = (vel.y + force.y) * damping;
    vel.z = (vel.z + force.z) * damping;

    // Update position
    pos_i.x += vel.x;
    pos_i.y += vel.y;
    pos_i.z += vel.z;

    // Store updated position and velocity
    nodes[idx].position[0] = pos_i.x;
    nodes[idx].position[1] = pos_i.y;
    nodes[idx].position[2] = pos_i.z;
    nodes[idx].velocity[0] = vel.x;
    nodes[idx].velocity[1] = vel.y;
    nodes[idx].velocity[2] = vel.z;

    // Flags and mass remain unchanged
}
